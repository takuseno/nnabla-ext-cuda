#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/one_hot.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename TI, typename T>
__global__ void kernel_one_hot(const int num, const int size, const TI *x, T *y,
                               const int *shape, const int dim) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    int addr = 0;
    Size_t s = 1;
    for (int i = dim - 1; i >= 0; --i) {
      addr += x[idx * dim + i] * s;
      s *= shape[i];
    }
    y[idx * size + addr] = (T)1;
  }
}

template <typename TI, typename T>
void OneHotCuda<TI, T>::setup_impl(const Variables &inputs,
                                   const Variables &outputs) {
  cuda_set_device(this->device_);
  OneHot<TI, T>::setup_impl(inputs, outputs);
}

template <typename TI, typename T>
void OneHotCuda<TI, T>::forward_impl(const Variables &inputs,
                                     const Variables &outputs) {
  cuda_set_device(this->device_);
  const TIcu *x = inputs[0]->get_data_pointer<TIcu>(this->ctx_);
  Tcu *y = outputs[0]->cast_data_and_get_pointer<Tcu>(this->ctx_, true);

  int *shape_cpu = new int[this->shape_.size()];
  for (int i = 0; i < this->shape_.size(); ++i) {
    shape_cpu[i] = this->shape_[i];
  }

  CudaCachedArray cushape(sizeof(int) * this->shape_.size(), dtypes::BYTE,
                          this->ctx_);
  void *cushape_ptr = cushape.pointer<void>();
  NBLA_CUDA_CHECK(hipMemcpy((int *)cushape_ptr, shape_cpu,
                             sizeof(int) * this->shape_.size(),
                             hipMemcpyHostToDevice));
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_one_hot, this->num_, this->size_, x, y,
                                 (int *)cushape_ptr, this->dim_);

  delete[] shape_cpu;
}
}
